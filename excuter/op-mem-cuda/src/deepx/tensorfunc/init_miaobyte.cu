#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "init_miaobyte.hpp"
#include "deepx/tensor.hpp"
#include "authors.hpp"
#include <hip/hip_fp16.h>

namespace deepx::tensorfunc
{
    template <typename T>
    __global__ void kernel_constant(T *data, int size, T value)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            data[idx] = value;
        }
    }



    // 实现特化版本的成员函数
    void _constant_func<miaobyte, float>::func(Tensor<float> &tensor, const float value)
    {
        int size = tensor.shape.size;
        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        
        kernel_constant<<<numBlocks, blockSize>>>(tensor.data, size, value);
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to launch constant kernel");
        }
    }

    void _constant_func<miaobyte, double>::func(Tensor<double> &tensor, const double value)
    {
        int size = tensor.shape.size;
        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        
        kernel_constant<<<numBlocks, blockSize>>>(tensor.data, size, value);
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to launch constant kernel");
        }
    }

    void _constant_func<miaobyte, __half>::func(Tensor<__half> &tensor, const __half value)
    {
        int size = tensor.shape.size;
        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        
        kernel_constant<<<numBlocks, blockSize>>>(tensor.data, size, value);
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to launch constant kernel");
        }
    }

        // 添加kernel函数
    template <typename T>
    __global__ void kernel_arange(T *data, int size, T start, T step)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            data[idx] = start + step * static_cast<T>(idx);
        }
    }

    void _arange_func<miaobyte, float>::func(Tensor<float> &tensor, const float start, const float step)
    {
        int size = tensor.shape.size;
        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        
        kernel_arange<<<numBlocks, blockSize>>>(tensor.data, size, start, step);
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to launch arange kernel");
        }
    }

    void _arange_func<miaobyte, double>::func(Tensor<double> &tensor, const double start, const double step)
    {
        int size = tensor.shape.size;
        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        
        kernel_arange<<<numBlocks, blockSize>>>(tensor.data, size, start, step);
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to launch arange kernel");
        }
    }

    void _arange_func<miaobyte, __half>::func(Tensor<__half> &tensor, const __half start, const __half step)
    {
        int size = tensor.shape.size;
        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        
        kernel_arange<<<numBlocks, blockSize>>>(tensor.data, size, start, step);
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to launch arange kernel");
        }
    }
}